#include "hip/hip_runtime.h"

#include <hipblas.h>

// clang-format off
#include "tensors/gpu/prod.h"
#include "tensors/gpu/backend.h"
#include "tensors/gpu/cuda_helpers.h"
// clang-format on

namespace marian {

namespace gpu {

void Prod(marian::Tensor C,
          marian::Tensor A,
          marian::Tensor B,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  hipSetDevice(C->getDevice().no);
  float alpha = scalar;

  size_t m = A->shape().elements() / A->shape().back();
  size_t k = A->shape().back();
  if(transA)
    std::swap(m, k);

  size_t l = B->shape().elements() / B->shape().back();
  size_t n = B->shape().back();
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape().back();
  size_t ldb = B->shape().back();
  size_t ldc = B->shape().back();

  if(transB)
    ldc = B->shape().elements() / B->shape().back();

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();

#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif

  hipblasSgemm(cublasHandle,
              opB,
              opA,
              n,
              m,
              k,
              &alpha,
              B->data(),
              ldb,
              A->data(),
              lda,
              &beta,
              C->data(),
              ldc);
#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif
}

__global__ void gAddBias(float* out, const float* bias, size_t length, size_t cols) {
  for(int bid = 0; bid < length; bid += blockDim.x * gridDim.x) {
    int index = bid + blockDim.x * blockIdx.x + threadIdx.x;
    if(index < length) {
      size_t index2 = index % cols;
      out[index] += bias[index2];
    }
  }
}

void AddBias(marian::Tensor C, const marian::Tensor bias) {
  hipSetDevice(C->getDevice().no);

  int length = C->shape().elements();
  int cols = bias->shape().elements();

  int threads = std::min(MAX_THREADS, length);
  int blocks = std::min(MAX_BLOCKS, length / threads + (length % threads != 0));

  gAddBias<<<blocks, threads>>>(C->data(), bias->data(), length, cols);

  hipStreamSynchronize(0);
}

void ProdWithBias(marian::Tensor C,
          const marian::Tensor A,
          const marian::Tensor B,
          const marian::Tensor bias,
          bool transA,
          bool transB,
          float beta,
          float scalar) {
  marian::gpu::Prod(C, A, B, transA, transB, beta, scalar);
  marian::gpu::AddBias(C, bias);
}

void ProdBatched(marian::Tensor C,
                 Ptr<Allocator> allocator,
                 const marian::Tensor A,
                 const marian::Tensor B,
                 bool transA,
                 bool transB,
                 float beta,
                 float scalar) {
  hipSetDevice(C->getDevice().no);
  float alpha = scalar;

  size_t batchA = A->shape().elements() / (A->shape()[-1] * A->shape()[-2]);
  size_t batchB = B->shape().elements() / (B->shape()[-1] * B->shape()[-2]);

  size_t m = A->shape()[-2];
  size_t k = A->shape()[-1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[-2];
  size_t n = B->shape()[-1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[-1];
  size_t ldb = B->shape()[-1];
  size_t ldc = B->shape()[-1];

  if(transB)
    ldc = B->shape()[-2];

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();


  int strideA = batchA == 1 ? 0 : m * k;
  int strideB = batchB == 1 ? 0 : n * k;
  int strideC = n * m;
  int batchC = std::max(batchA, batchB);

  std::vector<const float*> aptr;
  std::vector<const float*> bptr;
  std::vector<float*> cptr;

  for(int i = 0; i < batchC; i++) {
    aptr.push_back(A->data() + (i % batchA) * strideA);
    bptr.push_back(B->data() + (i % batchB) * strideB);
    cptr.push_back(C->data() + i * strideC);
  }

  auto mp_aptr = allocator->alloc<const float*>(aptr.size());
  CudaCopy(aptr.data(), aptr.data() + aptr.size(), mp_aptr->data<const float*>());

  auto mp_bptr = allocator->alloc<const float*>(bptr.size());
  CudaCopy(bptr.data(), bptr.data() + bptr.size(), mp_bptr->data<const float*>());

  auto mp_cptr = allocator->alloc<float*>(cptr.size());
  CudaCopy(cptr.data(), cptr.data() + cptr.size(), mp_cptr->data<float*>());

#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif
  hipblasSgemmBatched(cublasHandle,
                      opB,
                      opA,
                      n,
                      m,
                      k,
                      &alpha,
                      mp_bptr->data<const float*>(),
                      ldb,
                      mp_aptr->data<const float*>(),
                      lda,
                      &beta,
                      mp_cptr->data<float*>(),
                      ldc,
                      batchC);
#if CUDA_VERSION >= 9000
  hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif

  allocator->free(mp_aptr);
  allocator->free(mp_bptr);
  allocator->free(mp_cptr);
}

/*
void ProdBatched(marian::Tensor C,
                 const marian::Tensor A,
                 const marian::Tensor B,
                 const std::vector<size_t>& indicesA,
                 const std::vector<size_t>& indicesB,
                 bool transA,
                 bool transB,
                 float beta,
                 float scalar) {
  hipSetDevice(C->getDevice().no);
  float alpha = scalar;

  size_t batchA = A->shape().elements() / (A->shape()[-1] * A->shape()[-2]);
  size_t batchB = B->shape().elements() / (B->shape()[-1] * B->shape()[-2]);
  const size_t batchCount = std::max(batchA, batchB);

  size_t m = A->shape()[-2];
  size_t k = A->shape()[-1];
  if(transA)
    std::swap(m, k);

  size_t l = B->shape()[-2];
  size_t n = B->shape()[-1];
  if(transB)
    std::swap(l, n);

  size_t lda = A->shape()[-1];
  size_t ldb = B->shape()[-1];
  size_t ldc = B->shape()[-1];

  if(transB)
    ldc = B->shape()[-2];

  float **a_ptrs_host, **b_ptrs_host, **c_ptrs_host;
  a_ptrs_host = new float*[batchA];
  b_ptrs_host = new float*[batchB];
  c_ptrs_host = new float*[batchCount];
  for(int i = 0; i < std::max(batchA, batchB); ++i) {
    a_ptrs_host[i] = A->data() + (indicesA.size() > 0 ? indicesA[i] * m * k : 0);
    b_ptrs_host[i] = B->data() + (indicesB.size() > 0 ? indicesB[i] * l * n : 0);
    // c_ptrs_host[i] = C->data() + i * n * m;
    c_ptrs_host[i] = (i > 3 ? C->data() + i * n * m : C->data());
  }

  float **a_ptrs, **b_ptrs, **c_ptrs;
  CUDA_CHECK(hipMalloc((void**)&a_ptrs, batchA * sizeof(float*)));
  CUDA_CHECK(hipMalloc((void**)&b_ptrs, batchB * sizeof(float*)));
  CUDA_CHECK(hipMalloc((void**)&c_ptrs, batchCount * sizeof(float*)));
  hipMemcpy(a_ptrs, a_ptrs_host, batchA * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(b_ptrs, b_ptrs_host, batchB * sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(c_ptrs, c_ptrs_host, batchCount * sizeof(float*), hipMemcpyHostToDevice);

  hipblasOperation_t opA = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t opB = transB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  auto cublasHandle = std::static_pointer_cast<gpu::Backend>(C->getBackend())
                          ->getCublasHandle();

#if CUDA_VERSION >= 9000
// hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);
#endif
  hipblasSgemmBatched(cublasHandle,
                     opB,
                     opA,
                     n,
                     m,
                     k,
                     &alpha,
                     (const float**)b_ptrs, // TODO Why do I have to explicitly cast to const?
                     ldb,
                     (const float**)a_ptrs, // TODO - compiler warning otherwise
                     lda,
                     &beta,
                     c_ptrs, // NOTE
                     ldc,
                     batchCount);
#if CUDA_VERSION >= 9000
// hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH);
#endif

  CUDA_CHECK(hipFree(a_ptrs));
  CUDA_CHECK(hipFree(b_ptrs));
  CUDA_CHECK(hipFree(c_ptrs));
}
*/
}
}
